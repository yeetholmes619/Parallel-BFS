#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include<iomanip>
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include<chrono>
#include <math.h>
#include<limits.h>
#define NUM_NODES 50'000'005
#define NUM_EDGES 300'000'005

using namespace std;
typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

Node node[NUM_NODES];
int edges[NUM_EDGES];
bool curr_frontier[NUM_NODES] = { false }, next_frontier[NUM_NODES] = { false };
bool visited[NUM_NODES] = { false };
int source = 1;
int num_nodes, num_edges;
int degree[NUM_NODES];
const int alpha = 14;
const int beta = 24;

//pointers
int *num_nodes_ptr;
Node* Va;
int* Ea;
bool* Cf;
bool* Nf;
bool* Xa;
bool* done;
unsigned int *nf_ptr;
unsigned int *mf_ptr;
unsigned int *m_unvisited_ptr;
int* degree_ptr;
int* num_edges_ptr;
int n_visited = 0;

//Xa -> Visited
//Ea -> CSR edges
//Va -> CSR start and lengths
//Fa -> Frontier


__device__ bool valid_idx(int idx,int *num_nodes_ptr){
    return idx < *num_nodes_ptr;
}

__device__ void next_true(int idx,bool* Nf,unsigned int* nf_ptr,unsigned int* mf_ptr,int* degree_ptr){
	Nf[idx] = true;
	atomicInc(nf_ptr,INT_MAX);
	atomicAdd(mf_ptr,degree_ptr[idx]);

}

__device__ void current_false(int idx,unsigned int* nf_ptr,unsigned int* mf_ptr,int* degree_ptr,bool* Cf){
	Cf[idx] = false;
	atomicDec(nf_ptr,INT_MAX);
	atomicSub(mf_ptr,degree_ptr[idx]);
}

__device__ void visit(int idx,unsigned int*m_unvisited_ptr,int* degree_ptr,bool* Xa){
	Xa[idx] = true;
	atomicSub(m_unvisited_ptr,degree_ptr[idx]);
}
	
__global__ void TOPDOWN_BFS_KERNEL(Node *Va, int *Ea, bool *Cf, bool *Nf, bool *Xa, bool *done,unsigned int*nf_ptr,unsigned int*mf_ptr,unsigned int* m_unvisited_ptr,int*num_nodes_ptr,int* degree_ptr)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;


	if ( valid_idx(id,num_nodes_ptr) && Cf[id] == true )
	{
		//printf("%d ", id); //This printf gives the order of vertices in BFS	
		visit(id,m_unvisited_ptr,degree_ptr,Xa);
		current_false(id,nf_ptr,mf_ptr,degree_ptr,Cf);
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++) 
		{
			int nid = Ea[i];

			if (Cf[nid] == false and Xa[nid] == false)
			{
				next_true(nid,Nf,nf_ptr,mf_ptr,degree_ptr);
				*done = false;
			}

		}

	}



}

__global__ void BOTTOMUP_BFS_KERNEL(Node *Va, int *Ea, bool *Cf, bool *Nf, bool *Xa, bool *done,unsigned int*nf_ptr,unsigned int*mf_ptr,unsigned int*
		 m_unvisited_ptr,int*num_nodes_ptr,int* degree_ptr)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;


	if ( valid_idx(id,num_nodes_ptr) && Xa[id] == false)
	{
		if(Cf[id] == true){
			//printf("%d ", id); //This printf gives the order of vertices in BFS	
			visit(id,m_unvisited_ptr,degree_ptr,Xa);
		}else{
			int start = Va[id].start;
			int end = start + Va[id].length;
			for (int i = start; i < end; i++) 
			{
				int nid = Ea[i];

				if (Cf[nid] == true)
				{
					next_true(id,Nf,nf_ptr,mf_ptr,degree_ptr);
					*done = false;
					break;
				}

			}
		}
	}

	__syncthreads();
    if(valid_idx(id,num_nodes_ptr))
	    current_false(id,nf_ptr,mf_ptr,degree_ptr,Cf);

}




int num_blks;
int threads;
int state; //0 -> TOPDOWN, 1 -> BOTTOMUP
//TODO : account for increase or decrease in nodes and edges in frontier
__global__ void swapPointers(bool** a, bool** b){
	bool* temp = *a;
	*a = *b;
	*b = temp;
}

int n_top_down = 0;
int n_bottom_up = 0;

void BFS_BOILERPLATE()
{  


    	auto start_pp = std::chrono::high_resolution_clock::now();
	int nf = 1;
	int mf = degree[source];
	int m_unvisited = num_edges;
	bool done_val = true;
	curr_frontier[source] = true;

	hipMalloc((void**)&num_nodes_ptr, sizeof(int));
	hipMemcpy(num_nodes_ptr, &num_nodes, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void**)&nf_ptr, sizeof(unsigned int));
	hipMemcpy(nf_ptr,&nf,sizeof(unsigned int),hipMemcpyHostToDevice);

        hipMalloc((void**)&mf_ptr, sizeof(unsigned int));
	hipMemcpy(mf_ptr,&mf,sizeof(unsigned int),hipMemcpyHostToDevice);

        hipMalloc((void**)&m_unvisited_ptr, sizeof(unsigned int));
	hipMemcpy(m_unvisited_ptr,&num_edges,sizeof(unsigned int),hipMemcpyHostToDevice);

	hipMalloc((void**)&num_edges_ptr,sizeof(unsigned int));
	hipMemcpy(num_edges_ptr,&num_edges,sizeof(unsigned int),hipMemcpyHostToDevice);

        hipMalloc((void**)&degree_ptr, sizeof(int)*num_nodes);
        hipMemcpy(degree_ptr, degree, sizeof(int)*num_nodes, hipMemcpyHostToDevice);


        hipMalloc((void**)&Va, sizeof(Node)*num_nodes);
        hipMemcpy(Va, node, sizeof(Node)*num_nodes, hipMemcpyHostToDevice);

        hipMalloc((void**)&Ea, sizeof(int)*num_edges);
        hipMemcpy(Ea, edges, sizeof(int)*num_edges, hipMemcpyHostToDevice);

        hipMalloc((void**)&Cf, sizeof(bool)*num_nodes);
        hipMemcpy(Cf, curr_frontier, sizeof(bool)*num_nodes, hipMemcpyHostToDevice);


        hipMalloc((void**)&Nf, sizeof(bool)*num_nodes);
        hipMemcpy(Nf, next_frontier, sizeof(bool)*num_nodes, hipMemcpyHostToDevice);

        hipMalloc((void**)&Xa, sizeof(bool)*num_nodes);
        hipMemcpy(Xa, visited, sizeof(bool)*num_nodes, hipMemcpyHostToDevice);

        hipMalloc((void**)&done, sizeof(bool));


    	auto end_pp = std::chrono::high_resolution_clock::now();
    	std::chrono::duration<double, std::nano> duration_pp = end_pp - start_pp;
    	auto start_bfs = std::chrono::high_resolution_clock::now();
	do{
		done_val =true;
		hipMemcpy(done,&done_val,sizeof(bool),hipMemcpyHostToDevice);
		if(state == 0 and mf > m_unvisited/alpha){
			//printf("Going from top-down to bottom-up\n");
			state = 1;
		}
		else if(state == 1 and nf < num_nodes/beta){
			//printf("Going from bottom-up to top-down\n");
			state = 0;
		}

		if(state == 0){
			n_top_down++;	
			TOPDOWN_BFS_KERNEL<<<num_blks, threads>>>(Va, Ea,  Cf, Nf,  Xa,  done,nf_ptr,mf_ptr,m_unvisited_ptr,num_nodes_ptr, degree_ptr);
		}
		else{
			n_bottom_up++;	
			BOTTOMUP_BFS_KERNEL<<<num_blks, threads>>>(Va, Ea,  Cf, Nf,  Xa,  done,nf_ptr,mf_ptr,m_unvisited_ptr,num_nodes_ptr, degree_ptr);
		}

		hipMemcpy(&done_val, done, sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(&mf, mf_ptr, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&nf, nf_ptr, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&m_unvisited, m_unvisited_ptr, sizeof(int), hipMemcpyDeviceToHost);
		
		//swapping current and next frontiers..
		bool *tmp = Cf;
		Cf = Nf;
		Nf = tmp;
	}while(!done_val);
    	auto end_bfs = std::chrono::high_resolution_clock::now();
    	std::chrono::duration<double, std::nano> duration_bfs = end_bfs - start_bfs;
	cout<<fixed<<setprecision(12);
	cout<<num_nodes<<" "<<num_edges<<" "<<duration_pp.count()<<" "<<duration_bfs.count()<<"\n";

        //hipMemcpy(visited, Xa, sizeof(bool)*num_nodes, hipMemcpyDeviceToHost);
	//for(int i =0 ; i < num_nodes;i++) n_visited += visited[i];

	//TODO: write free for all pointers
        hipFree(Va);
        hipFree(Ea);
        hipFree(Cf);
        hipFree(Nf);
        hipFree(Xa);
        hipFree(done);
	hipFree(nf_ptr);
	hipFree(mf_ptr);
	hipFree(m_unvisited_ptr);
	hipFree(num_edges_ptr);
	hipFree(degree_ptr);

}

// The BFS frontier corresponds to all the nodes being processed at the current level.


int main()
{
    cin>>num_nodes>>num_edges;
    //num_blks,threads
    threads = min((1<<10), num_nodes);
    num_blks = (num_nodes + threads-1)/threads;

    for(int i = 0; i < num_nodes; i++){
            cin>>node[i].start>>node[i].length;
            degree[i] = node[i].length;
    }

    for(int i = 0; i < num_edges; i++){
            cin>>edges[i];
    }



	BFS_BOILERPLATE();



}

